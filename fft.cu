#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

//-----------------------------------------------------------------------------
__global__ void kernelFunc(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M, int j) 
{
	int k = bx * 512 + tx;
	
	float* X_r_d;
	float* X_i_d;
	
	if(j == N/2) {
		
		int i, p, q;
			
		p = 2*k;
		q = (p & 1) * N/2;
		
		for(i=1; i<M; i+=2) {
			q = q + ((p & (3 << i)) >> i) * (1 << M-i-2); 
		}
		
		X_r_d[q] = x_r_d[p] + x_r_d[p+1];
		X_i_d[q] = x_i_d[p] + x_i_d[p+1];
		X_r_d[q+N/2] = x_r_d[p] - x_r_d[p+1];
		X_i_d[q+N/2] = x_i_d[p] - x_i_d[p+1];
	
	} else {
		int i, m, n;
		float z_r[4], z_i[4], w_r[4], w_i[4], temp_r[4], temp_i[4];

		n = (k/(N/(4*j)))*(N/j) + (k%(N/(4*j)));
		
		for(i=0; i<4; i++){
			
			temp_r[i] = x_r_d[n+i*N/(4*j)];
			temp_i[i] = x_i_d[n+i*N/(4*j)];
			
			m = (j==N/4)?0:(k%(N/(4*j)))*i*j;
			
			w_r[i] =  cos((2*PI*m)/N);
			w_i[i] = -sin((2*PI*m)/N);
			
		}

		z_r[0] = temp_r[0] + temp_r[1] + temp_r[2] + temp_r[3];
		z_i[0] = temp_i[0] + temp_i[1] + temp_i[2] + temp_i[3];

		z_r[1] = temp_r[0] + temp_i[1] - temp_r[2] - temp_i[3]; 
		z_i[1] = temp_i[0] - temp_r[1] - temp_i[2] + temp_r[3];
		
		z_r[2] = temp_r[0] - temp_r[1] + temp_r[2] - temp_r[3]; 
		z_i[2] = temp_i[0] - temp_i[1] + temp_i[2] - temp_i[3];
		
		z_r[3] = temp_r[0] - temp_i[1] - temp_r[2] + temp_i[3]; 
		z_i[3] = temp_i[0] + temp_r[1] - temp_i[2] - temp_r[3];
		
		for(i=0; i<4; i++){
			
			x_r_d[n+i*N/(4*j)] = w_r[i]*z_r[i] - w_i[i]*z_i[i];
			x_i_d[n+i*N/(4*j)] = w_r[i]*z_i[i] + w_i[i]*z_r[i];
			
		}
		
		if(j == N/4) {
			int p, q;
			
			p = 4*k;
			q = 0;
			
			for(i=0; i<M; i+=2) {
				q = q + ((p & (3 << i)) >> i) * (1 << M-i-2);
			}
			
			X_r_d[q] = x_r_d[p];
			X_i_d[q] = x_i_d[p];
			X_r_d[q+N/4] = x_r_d[p+1];
			X_i_d[q+N/4] = x_i_d[p+1];
			X_r_d[q+N/2] = x_r_d[p+2];
			X_i_d[q+N/2] = x_i_d[p+2];
			X_r_d[q+(3*N)/4] = x_r_d[p+3];
			X_i_d[q+(3*N)/4] = x_i_d[p+3];
		}
	}
	x_i_d = X_i_d;
	x_r_d = X_r_d;
}

//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, /*float* X_r_d, float* X_i_d,*/ const unsigned int N, const unsigned int M)
{
	int j;

	dim3 dimGrid(N/2048,1);
	dim3 dimBlock(512,1);

	for(j=1; j<N/2; j*=4) 
		kernelFunc <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, N, M, j);
	
	
	if(M % 2 == 1) {
		dim3 dimGrid(N/1024,1);
		dim3 dimBlock(512,1);	
		kernelFunc <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, N, M, N/2);
	}
	
}
